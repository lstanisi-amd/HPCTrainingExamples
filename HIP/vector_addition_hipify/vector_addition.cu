/*
Copyright (c) 2015-2023 Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/* Macro for checking GPU API return values */
#define gpuCheck(call)                                                                           \
do{                                                                                              \
    hipError_t gpuErr = call;                                                                   \
    if(hipSuccess != gpuErr){                                                                   \
        printf("GPU API Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(gpuErr)); \
        exit(1);                                                                                 \
    }                                                                                            \
}while(0)

/* --------------------------------------------------
Vector addition kernel
-------------------------------------------------- */
__global__ void vector_addition(double *A, double *B, double *C, int n)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < n) C[id] = A[id] + B[id];
}

/* --------------------------------------------------
Main program
-------------------------------------------------- */
int main(int argc, char *argv[]){

    /* Size of array */
    int N = 1024 * 1024;

    /* Bytes in array in double precision */
    size_t bytes = N * sizeof(double);

    /* Allocate memory for host arrays */
    double *h_A = (double*)malloc(bytes);
    double *h_B = (double*)malloc(bytes);
    double *h_C = (double*)malloc(bytes);

    /* Initialize host arrays */
    for(int i=0; i<N; i++){
        h_A[i] = sin(i) * sin(i); 
        h_B[i] = cos(i) * cos(i);
        h_C[i] = 0.0;
    }    

    /* Allocate memory for device arrays */
    double *d_A, *d_B, *d_C;
    gpuCheck( hipMalloc(&d_A, bytes) );
    gpuCheck( hipMalloc(&d_B, bytes) );
    gpuCheck( hipMalloc(&d_C, bytes) );

    /* Copy data from host arrays to device arrays */
    gpuCheck( hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice) );
    gpuCheck( hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice) );
    gpuCheck( hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice) );

    /* Set kernel configuration parameters
           thr_per_blk: number of threads per thread block
           blk_in_grid: number of thread blocks in grid */
    int thr_per_blk = 256;
    int blk_in_grid = ceil( float(N) / thr_per_blk );

    /* Launch vector addition kernel */
    vector_addition<<<blk_in_grid, thr_per_blk>>>(d_A, d_B, d_C, N);

    /* Check for kernel launch errors */
    gpuCheck( hipGetLastError() );

    /* Check for kernel execution errors */
    gpuCheck ( hipDeviceSynchronize() );

    /* Copy data from device array to host array (only need result, d_C) */
    gpuCheck( hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost) );

    /* Check for correct results */
    double sum       = 0.0;
    double tolerance = 1.0e-14;

    for(int i=0; i<N; i++){
        sum = sum + h_C[i];
    } 

    if( fabs( (sum / N) - 1.0 ) > tolerance ){
        printf("Error: Sum/N = %0.2f instead of ~1.0\n", sum / N);
        exit(1);
    }

    /* Free CPU memory */
    free(h_A);
    free(h_B);
    free(h_C);

    /* Free Device memory */
    gpuCheck( hipFree(d_A) );
    gpuCheck( hipFree(d_B) );
    gpuCheck( hipFree(d_C) );

    printf("\n==============================\n");
    printf("__SUCCESS__\n");
    printf("------------------------------\n");
    printf("N                : %d\n", N);
    printf("Blocks in Grid   : %d\n",  blk_in_grid);
    printf("Threads per Block: %d\n",  thr_per_blk);
    printf("==============================\n\n");

    return 0;
}
